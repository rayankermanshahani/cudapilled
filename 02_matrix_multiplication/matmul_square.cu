/* matmul_square.cu -- matrix mulitplication assuming square input matrices  */

#include <cmath>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define N 1024 /* square matrix width */
#define CUDA_CHECK(err) (cudaCheck(err, __FILE__, __LINE__))

/* function declarations */
void cudaCheck(hipError_t err, const char *file, int line);
void initMatrix(float *A, int n);
bool compareMatrices(const float *A, const float *B, int n, float tolerance);
template <typename KernelFunc>
bool verifyMatmulKernel(KernelFunc kernel, int n, float tolerance = 1e-4);
__global__ void matmulKernel(const float *A, const float *B, float *C, int n);

/* driver function */
int main(int argc, char **argv) {
  bool correct = verifyMatmulKernel(matmulKernel, N);

  fprintf(stdout, "MATRIX MULTIPLICATION PROGRAM COMPLETE\n");

  return correct ? 0 : 1;
}

/* cuda error handling */
void cudaCheck(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    fprintf(stderr, "[CUDA ERROR] at file %s:%d\n%s\n", file, line,
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

/* randonly initialize square matrix */
void initMatrix(float *A, int n) {
  for (int i = 0; i < n * n; ++i) {
    A[i] = static_cast<float>(rand()) / RAND_MAX;
  }
}

/* compare matrices with a tolerance */
bool compareMatrices(const float *A, const float *B, int n, float tolerance) {
  for (int i = 0; i < n * n; ++i) {
    if (std::fabs(A[i] - B[i]) > tolerance) {
      return false;
    }
  }
  return true;
}

/* verify cuda matmul kernel executed correctly */
template <typename KernelFunc>
bool verifyMatmulKernel(KernelFunc kernel, int n, float tolerance) {
  size_t size = n * n * sizeof(float);

  /* declare host matrices */
  float *A_h, *B_h, *C_h_cuda, *C_h_cublas;

  /* declare device matrices */
  float *A_d, *B_d, *C_d;

  /* allocate memory for host matrices */
  A_h = new float[n * n];
  B_h = new float[n * n];
  C_h_cuda = new float[n * n];
  C_h_cublas = new float[n * n];

  /* allocate memory for device matrices */
  CUDA_CHECK(hipMalloc((void **)&A_d, size));
  CUDA_CHECK(hipMalloc((void **)&B_d, size));
  CUDA_CHECK(hipMalloc((void **)&C_d, size));

  /* initalize host input matrices */
  initMatrix(A_h, n);
  initMatrix(B_h, n);

  /* copy host input matrices to device */
  CUDA_CHECK(hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice));

  /* launch config parameters */
  dim3 blockDim(16, 16, 1);
  dim3 gridDim(ceil(n / float(blockDim.x)), ceil(n / float(blockDim.y)), 1);

  /* launch matmul kernel */
  kernel<<<gridDim, blockDim>>>(A_d, B_d, C_d, n);
  CUDA_CHECK(hipDeviceSynchronize());

  /* copy result device matrix to host */
  CUDA_CHECK(hipMemcpy(C_h_cuda, C_d, size, hipMemcpyDeviceToHost));

  /* compute reference result via cuBLAS */
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float alpha = 1.0f;
  float beta = 0.0f;
  hipblasStatus_t status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n,
                                      &alpha, A_d, n, B_d, n, &beta, C_d, n);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "cuBLAS error: %d\n", status);
  }
  CUDA_CHECK(hipMemcpy(C_h_cublas, C_d, size, hipMemcpyDeviceToHost));
  hipblasDestroy(handle);

  /* compare results */
  bool correct = compareMatrices(C_h_cuda, C_h_cublas, n, tolerance);
  if (correct) {
    fprintf(stdout, "Success: CUDA kernel output matches cuBLAS result within "
                    "specified tolerance.\n");
  } else {
    fprintf(stdout,
            "Failure: CUDA kernel output does not match cuBLAS result.\n");
    for (int i = 0; i < 5; ++i) {
      fprintf(stdout, "C_h_cuda[%d] = %f, C_h_cublas[%d] = %f\n", i,
              C_h_cuda[i], i, C_h_cublas[i]);
    }
  }

  /* clean up memory */
  delete[] A_h;
  delete[] B_h;
  delete[] C_h_cuda;
  delete[] C_h_cublas;
  CUDA_CHECK(hipFree(A_d));
  CUDA_CHECK(hipFree(B_d));
  CUDA_CHECK(hipFree(C_d));

  return correct;
}

/* matrix multiplication: assuming that operand matrices are squares */
__global__ void matmulKernel(const float *A, const float *B, float *C, int n) {
  int row = blockIdx.y * blockDim.y + threadIdx.y; /* row in A */
  int col = blockIdx.x * blockDim.x + threadIdx.x; /* col in B */

  /* check if row and col values are valid */
  if ((row < n) && (col < n)) {
    float acc = 0.0f;
    /* compute inner dot product of A's row and B's col */
    for (int k = 0; k < n; ++k) {
      acc += A[row * n + k] * B[k * n + col];
    }
    C[row * n + col] = acc;
  }
}
