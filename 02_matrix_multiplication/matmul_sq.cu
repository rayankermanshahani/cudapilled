#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 1024 // square matrix width
#define THREADS_PER_BLOCK 256

#define CUDA_CHECK(err) cudaCheck(err, __FILE__, __LINE__)

/* function declarations */
__global__ void matMul(const float *A, const float *B, float *C, int n);
void cpuMatMul(const float *A, const float *B, float *C, int n);
void initRand(float *A, int n);
void cudaCheck(hipError_t err, const char *file, int line);

/* driver program */
int main(void) {
  srand(time(NULL));                   // initialize rng
  float *A_h, *B_h, *C_h, *C_cpu;      // host arrays
  float *A_d, *B_d, *C_d;              // device arrays
  size_t size = N * N * sizeof(float); // array size in bytes

  // allocate host memory
  A_h = (float *)malloc(size);
  B_h = (float *)malloc(size);
  C_h = (float *)malloc(size);
  C_cpu = (float *)malloc(size);

  // allocate device memory
  CUDA_CHECK(hipMalloc(&A_d, size));
  CUDA_CHECK(hipMalloc(&B_d, size));
  CUDA_CHECK(hipMalloc(&C_d, size));

  // init input host arrays
  printf("Initializing A and B on host...\n"); // TODO: diagnostic prints
  initRand(A_h, N);
  initRand(B_h, N);

  // copy input arrays from host to device
  printf("Copying A and B from host to device...\n"); // TODO: diagnostic prints
  CUDA_CHECK(hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice));

  // launch kernel for simple vector addition
  printf("Computing matmul on GPU...\n"); // TODO: diagnostic prints
  dim3 threadsPerBlock(THREADS_PER_BLOCK, 1, 1);
  dim3 blocksPerGrid(ceil(N / float(threadsPerBlock.x)), 1, 1);
  matMul<<<blocksPerGrid, threadsPerBlock>>>(A_d, B_d, C_d, N);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  printf("Matmul on GPU complete.\n"); // TODO: diagnostic prints

  // copy output array from device to host
  CUDA_CHECK(hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost));

  printf("Computing matmul on CPU...\n"); // TODO: diagnostic prints
  cpuMatMul(A_h, B_h, C_cpu, N);
  printf("Matmul on CPU complete.\n"); // TODO: diagnostic prints

  // verify result
  for (int i = 0; i < 5; ++i) {
    printf("C_gpu[%d] = %f, C_cpu[%d] = %f\n", i, C_h[i], i, C_cpu[i]);
  }

  // clean up memory
  printf("Cleaning up memory...\n"); // TODO: diagnostic prints
  free(A_h);
  free(B_h);
  free(C_h);
  free(C_cpu);
  CUDA_CHECK(hipFree(A_d));
  CUDA_CHECK(hipFree(B_d));
  CUDA_CHECK(hipFree(C_d));

  fprintf(stdout, "VECTOR ADDITION PROGRAM COMPLETE.\n");
  return 0;
}

/* matrix multiplication */
__global__ void matMul(const float *A, const float *B, float *C, int n) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < n && col < n) {
    float acc = 0.0f;
    for (int k = 0; k < n; ++k) {
      acc += A[row * n + k] * B[k * n + col];
    }
    C[row * n + col] = acc;
  }
}

/* matrix multiplication on the cpu */
void cpuMatMul(const float *A, const float *B, float *C, int n) {
  for (int i = 0; i < n; ++i) {     // i-th row of A
    for (int j = 0; j < n; ++j) {   // j-th col of B
      for (int k = 0; k < n; ++k) { // k-th element of corresponding row and col
        C[i * n + j] += A[i * n + k] + B[k * n + j];
      }
    }
  }
}

/* randomly initalize float array */
void initRand(float *a, int n) {
  for (int i = 0; i < n; ++i) {
    a[i] = (float)rand() / (float)RAND_MAX;
  }
}

/* cuda error handling */
void cudaCheck(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
