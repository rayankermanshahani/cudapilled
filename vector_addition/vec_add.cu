/* simple vector addition */
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <time.h>

#define N 1000000 /* number of elements in each array (object) */

/* cuda error handling */
void cudaCheck(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    fprintf(stderr, "[CUDA ERROR] at file %s:%d\n%s\n", file, line,
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
#define CUDA_CHECK(err) (cudaCheck(err, __FILE__, __LINE__))

/* randomly initialize array */
void rand_init(float *A, long int n) {
  for (long int i = 0; i < n; ++i) {
    A[i] = (float)rand() / RAND_MAX;
  }
}

/* initialize array with a given float */
void val_init(float *A, long int n, float value) {
  for (long int i = 0; i < n; ++i) {
    A[i] = value;
  }
}

/* vector addition kernel */
__global__ void vecAddKernel(const float *A, const float *B, float *C,
                             long int n) {
  long int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

int main(void) {
  /* monotonically time the operation */
  struct timespec start_time, end_time;

  /* size of each object in bytes */
  long int size = N * sizeof(float);

  /* declare objects on host (CPU) */
  float *A_h, *B_h, *C_h;

  /* declare objects on device (GPU) */
  float *A_d, *B_d, *C_d;

  /* allocate memory for objects on host */
  A_h = (float *)malloc(size);
  B_h = (float *)malloc(size);
  C_h = (float *)malloc(size);

  /* allocate memory for objects on device */
  CUDA_CHECK(hipMalloc((void **)&A_d, size));
  CUDA_CHECK(hipMalloc((void **)&B_d, size));
  CUDA_CHECK(hipMalloc((void **)&C_d, size));

  /* initialize host objects */
  /*
  rand_init(A_h, N);
  rand_init(B_h, N);
  */
  val_init(A_h, N, 1.0);
  val_init(B_h, N, 2.0);

  /* copy operand objects from host to device */
  CUDA_CHECK(hipMemcpy(A_d, A_h, N, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(B_d, B_h, N, hipMemcpyHostToDevice));

  /* launch kernel with: N/256 blocks per grid and 256 threads per block */
  clock_gettime(CLOCK_MONOTONIC, &start_time);
  vecAddKernel<<<ceil(N / 256.0), 256>>>(A_d, B_d, C_d, N);
  clock_gettime(CLOCK_MONOTONIC, &end_time);

  double elapsed_time = (end_time.tv_sec - start_time.tv_sec) +
                        (end_time.tv_nsec - start_time.tv_nsec) / 1e9;

  /* copy result object from device to host */
  CUDA_CHECK(hipMemcpy(C_h, C_d, N, hipMemcpyDeviceToHost));

  /* free host memory */
  free(A_h);
  free(B_h);
  free(C_h);

  /* free device memory */
  CUDA_CHECK(hipFree(A_d));
  CUDA_CHECK(hipFree(B_d));
  CUDA_CHECK(hipFree(C_d));

  fprintf(stdout,
          "It took %f seconds to add two vectors of length %d in cuda.\n",
          elapsed_time, N);

  fprintf(stdout, "PROGRAM COMPLETE.\n");
  return 0;
}
